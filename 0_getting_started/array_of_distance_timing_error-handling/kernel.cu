#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdio.h>

#define M 100 // number of times to do the data transfer
#define TPB 32


__device__ float distance(float x1, float x2)
{
	return sqrt((x2 - x1) *(x2 - x1));
}

__global__ void distanceKernel(float *d_out, float *d_in, float ref)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const float x = d_in[i];
	d_out[i] = distance(x, ref);
	printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
}

// wrapper or launcher
void distanceArray(float *out, float *in, float ref, int len)
{
	// Create event variables for timing
	hipEvent_t startMemcpy, stopMemcpy;
	hipEvent_t startKernel, stopKernel;
	hipEventCreate(&startMemcpy);
	hipEventCreate(&stopMemcpy);
	hipEventCreate(&startKernel);
	hipEventCreate(&stopKernel);

	float *d_in = 0;
	float *d_out = 0;

	hipError_t err = hipMalloc(&d_in, len*sizeof(float));
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

	err = hipMalloc(&d_out, len*sizeof(float));
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));

	hipEventRecord(startMemcpy);
	// copy input data from host to device M times
	for (int i = 0; i < M; ++i)
	{
		hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);
	}

	hipEventRecord(stopMemcpy);

	hipEventRecord(startKernel);
	distanceKernel<<<len/TPB, TPB>>>(d_out, d_in, ref);
	hipEventRecord(stopKernel);
	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();

	// info on kernel launch errors
	if (errSync != hipSuccess) printf("%s\n", hipGetErrorString(errSync));

	// info on kernel execution errors
	if (errAsync != hipSuccess) printf("%s\n", hipGetErrorString(errAsync));

	err= hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));


	// Ensure timed events have stopped.
	hipEventSynchronize(stopMemcpy);
	hipEventSynchronize(stopKernel);

	// Convert event records to time and output
	float memcpyTimeInMs = 0;
	hipEventElapsedTime(&memcpyTimeInMs, startMemcpy, stopMemcpy);
	float kernelTimeInMs = 0;
	hipEventElapsedTime(&kernelTimeInMs, startKernel, stopKernel);	

	printf("kernel time (ms): %f\n", kernelTimeInMs);
	printf("data transfer time (ms): %f\n", memcpyTimeInMs);	

	hipFree(d_in);
	hipFree(d_out);

}